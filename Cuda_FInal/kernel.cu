#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <fstream>
#include <iostream>
#include <stdio.h>
#include <chrono>
#include<string>
#include <bitset>
#include <thrust/sequence.h>
#include <thrust/gather.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <chrono>

#pragma once
#include "HammingVectors.h"
#include "scans.cuh"
#include "HammingCPUVec.h"

#define ThreadNr 1024


using namespace std;
using namespace std::chrono;


hipError_t findPairs(HammingVectors* vec,int verbose);

void PrintVector(uint32_t* vec, int n, int l) {
    for (int i = 0; i < n; i++) {
        cout << endl;
        for (int j = 0; j < l; j++)
            cout << vec[i + (j * n)];
    }
}

//Wczytywanie Danych
bool LoadSequences(string path, HammingVectors*& vec)
{
    ifstream myfile(path, ifstream::binary);

    if (!myfile.is_open())
    {
        cout << "Unable to open file " << path << endl;
        myfile.close();
        return false;
    }

    int headerData[2];
    char no;

    myfile >> headerData[0];
    myfile >> no;
    myfile >> headerData[1];

    if (nullptr != vec)
        delete vec;

    vec = new HammingVectors(headerData[0], headerData[1]);
    char c;
    for (int i = 0; i < vec->vector_count; i++)
    {
        for (int j = 0; j < vec->vector_len; j++)
        {
            myfile >> c;
            vec->data[i + (j * vec->vector_count)] = c - '0';
            vec->invertedData[(vec->vector_len - 1 - j) * vec->vector_count + i] = c - '0';
        }

    }


    myfile.close();
    return true;
}




int main(int argc, char** argv)
{

    int cpu = 0;
    int ver = 1;
    /*
    if (argc < 2)
    {
        printf("Provide file with data\n");
        return 1;
    }

    if (argc == 3)
    {
        cpu = 1;
    }
    */
    

    //Fast IO initialization
    ios_base::sync_with_stdio(false);
    cin.tie(NULL);
    cout.tie(NULL);
    //Reading input data

    HammingVectors* hamSet = nullptr;

    cout << "Wczytywanie danych" << endl;
    auto start = high_resolution_clock::now();


    if (!LoadSequences("input.txt", hamSet))
        return;

    auto stop = high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds = stop - start;
    cout << "Wczytywanie ukonczone. Time: " << elapsed_seconds.count() << " [s]" << endl;

    findPairs(hamSet,ver);
    cout << endl;

    if (cpu)
    {
        HammingCPUVec cpuHam = HammingCPUVec(argv[1]);
        cout << "CPU Hamming" << endl;
        auto start = high_resolution_clock::now();
        cout << "Pairs found: " << cpuHam.hammingWithCPU(0) << endl;
        auto stop = high_resolution_clock::now();
        std::chrono::duration<double> elapsed_seconds = stop - start;

        cout << "Time: " << elapsed_seconds.count() << "[s]" << endl;
    }


    return 0;
}

//Raddix sort
void raddixSort(uint32_t* data, uint32_t* perm, uint32_t* dev_row, int n, int l)
{
    thrust::sequence(thrust::device, perm, perm + n, 0u);

    for (int i = l - 1; i >= 0; i--)
    {
        thrust::gather(thrust::device, perm, perm + n, data + i * n, dev_row);

        thrust::stable_sort_by_key(thrust::device, dev_row, dev_row + n, perm);
    }

    for (int i = 0; i < l; i++)
    {
        thrust::gather(thrust::device, perm, perm + n, data + i * n, dev_row);

        thrust::copy(thrust::device, dev_row, dev_row + n, data + i * n);
    }
}

//XOR Kernel
__global__ void xorKenrel(uint32_t* data, uint32_t* tmp, int n, int l)
{
    int idX = threadIdx.x + blockIdx.x * blockDim.x;
    if (idX < n - 1)
    {
        for (int i = 0; i < l; i++)
        {
            tmp[idX + n * i] = data[idX + n * i] ^ data[idX + 1 + n * i];
        }
    }
}

//XOR function
hipError_t xorVectors(uint32_t* data, uint32_t* dataInv, int n, int l)
{
    uint32_t* tmp = nullptr;
    hipError_t cudaStatus;

    cudaStatus=hipMalloc((void**)&tmp, n * l * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        goto ErrorXor;
    }

    int blockNr = n / ThreadNr;
    if (n % ThreadNr != 0)
        blockNr++;

    thrust::fill(thrust::device, tmp, tmp + n * l, 0u);
    xorKenrel <<<blockNr, ThreadNr >> > (data, tmp, n, l);
    thrust::copy(thrust::device, tmp, tmp + n * l, data);

    thrust::fill(thrust::device, tmp, tmp + n * l, 0u);
    xorKenrel <<<blockNr, ThreadNr >> > (dataInv, tmp, n, l);
    thrust::copy(thrust::device, tmp, tmp + n * l, dataInv);

    

ErrorXor:
    hipFree(tmp);
    
    return cudaStatus;

}

// Prefix scan with or operator
__global__ void  exclusiveOrKernel(uint32_t* data, int n, int l)
{
    uint32_t tmp;
    uint32_t tmp2;
    int idX = threadIdx.x + blockIdx.x * blockDim.x;

    if (idX < n)
    {

        tmp = data[idX + n];
        tmp2 = data[idX];
        data[idX + n] = data[idX];
        data[idX] = 0;


        for (int i = 2; i < l; i++)
        {
            if (tmp2 || tmp)
            {
                tmp2 = 1u;
                tmp = 1u;

                data[idX + n * i] = 1u;
            }
            else
            {
                tmp2 = tmp;
                tmp = data[idX + n * i];
                data[idX + n * i] = 0u;
            }

        }

    }

}


//Ustawianie tablic do szybkiego tworzenia finalnej tablicy
void setRightOrder(uint32_t* data, uint32_t* dev_perm, uint32_t* dev_row, int n, int l)
{
    thrust::sequence(thrust::device, dev_row, dev_row + n, 0u);

    thrust::stable_sort_by_key(thrust::device, dev_perm, dev_perm + n, dev_row);

    for (int i = 0; i < l; i++)
    {
        thrust::gather(thrust::device, dev_row, dev_row + n, data + i * n, dev_perm);

        thrust::copy(thrust::device, dev_perm, dev_perm + n, data + i * n);
    }

    thrust::copy(thrust::device, dev_row, dev_row + n, dev_perm);

}

//Tworzenie finalnej tablicy
void create_table(uint32_t* data, uint32_t* data_inv, uint32_t* dev_final, uint32_t* dev_row, int n, int l)
{
    thrust::copy(thrust::device, data, data + n * l, dev_final + l * n);

    for (int i = 0; i < l; i++)
    {
        thrust::copy(thrust::device, data_inv + (l - i - 1) * n, data_inv + (l - i) * n, (dev_final + n * l * 2) + (i * n));
        thrust::fill(thrust::device, dev_final + i * n, dev_final + (i + 1) * n, i);
    }
}

//Zliczanie Par
__global__ void countPairsKernel(uint32_t* data, uint64_t* pairs_count, int n, int l)
{
    int idX = blockIdx.x * blockDim.x + threadIdx.x;
    if (idX < n * l - 1)
        if (data[idX] == data[idX + 1] && data[idX + n * l] == data[idX + 1 + n * l] && data[idX + n * l * 2] == data[idX + 1 + n * l * 2])
            atomicAdd(pairs_count, 1);
}

__global__ void countPairsKernelVerbose(uint32_t* data, uint64_t* pairs_count, uint32_t* dev_perm, uint32_t* pairsOne, uint32_t* pairsTwo, int n, int l)
{
    int idX = blockIdx.x * blockDim.x + threadIdx.x;
    if (idX < (n * l) - 1)
        if (data[idX] == data[idX + 1] && data[idX + n * l] == data[idX + 1 + n * l] && data[idX + n * l * 2] == data[idX + 1 + n * l * 2])
        {
            
            if (idX< 1000 && idX<n*l-1)
            {
                int k = idX / n;
                pairsOne[idX] = (dev_perm[idX]-n*k);
                pairsTwo[idX] = (dev_perm[idX  + 1] - n*k);
                printf("Pairs %d\n", idX);
                
            }
            atomicAdd(pairs_count, 1);
        }
            
}

hipError_t findPairs(HammingVectors* vec,int verbose)
{
    uint32_t* dev_vec = nullptr;
    uint32_t* dev_invVec = nullptr;


    uint32_t* dev_row = nullptr;
    uint32_t* dev_perm = nullptr;
    uint32_t* dev_invPerm = nullptr;

    uint32_t* dev_finalTable = nullptr;

    uint64_t* dev_pair_count = nullptr;


    hipError_t cudaStatus;
    hipEvent_t start, stop;

    float time = 0, time_temp;
    uint64_t pairs;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    //Alokacja pamieci na wektory    
    cudaStatus = hipMalloc((void**)&dev_vec, vec->vector_len * vec->vector_count * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_vec failed!");
        goto Error;
    }
    //Alokacja pamieci na odwrócone wektory
    cudaStatus = hipMalloc((void**)&dev_invVec, vec->vector_len * vec->vector_count * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_invVec failed!");
        goto Error;
    }
   //pomocniczy wektor
    cudaStatus = hipMalloc((void**)&dev_row, vec->vector_count * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_row failed!");
        goto Error;
    }
    //Wektor permutacji
    cudaStatus = hipMalloc((void**)&dev_perm, vec->vector_count * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_perm failed!");
        goto Error;
    }
    //Wektor permutacji dla wektorów odróconych
    cudaStatus = hipMalloc((void**)&dev_invPerm, vec->vector_count * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_invPerm failed!");
        goto Error;
    }

    //finalna tablica
    cudaStatus = hipMalloc((void**)&dev_finalTable, vec->vector_count * vec->vector_len * 3 * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_finalTable failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_pair_count, sizeof(uint64_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_pair_count failed!");
        goto Error;
    }

    cudaStatus = hipMemset(dev_pair_count, 0, sizeof(uint64_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMamset dev_pair_count failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_vec, vec->data, vec->vector_len * vec->vector_count * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_invVec, vec->invertedData, vec->vector_len * vec->vector_count * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    cout << "Cuda start" << endl;

    int blockNr = vec->vector_count / ThreadNr;
    if (vec->vector_count % ThreadNr != 0)
        blockNr++;

    hipEventRecord(start, 0);

    //Sortowanie
    raddixSort(dev_vec, dev_perm, dev_row, vec->vector_count, vec->vector_len);
    raddixSort(dev_invVec, dev_invPerm, dev_row, vec->vector_count, vec->vector_len);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_temp, start, stop);
    time += time_temp;
    printf("Sorting time %f [s]\n", time_temp / 1000);

    cudaStatus = hipMemcpy(vec->data, dev_vec, vec->vector_len * vec->vector_count * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    bool equal = false;
    for (int i = 0; i < vec->vector_count-1; i++)
    {     
        for (int j = 0; j < vec->vector_len; j++)
        {
            if (vec->data[j*vec->vector_count + i] == vec->data[j *vec->vector_count + i+1])
            {
                equal = true;

            }
            else
            {
                equal = false;
                break;
            }
        }
        if (equal)
        {
            printf("Wektory powtarzalne wynik nieprawdziwy!\n");
            break;
        }
            
    }


    hipEventRecord(start, 0);

    //XOR
    cudaStatus=xorVectors(dev_vec, dev_invVec, vec->vector_count, vec->vector_len);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc XOR failed!");
        goto Error;
    }


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_temp, start, stop);
    time += time_temp;
    printf("XOR time %f [s]\n", time_temp / 1000);



    hipEventRecord(start, 0);

    //Prefix OR
    exclusiveOrKernel << <blockNr, ThreadNr >> > (dev_vec, vec->vector_count, vec->vector_len);
    exclusiveOrKernel << <blockNr, ThreadNr >> > (dev_invVec, vec->vector_count, vec->vector_len);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_temp, start, stop);
    time += time_temp;
    printf("Exclusive OR time %f [s]\n", time_temp / 1000);

    hipEventRecord(start, 0);


    //Prefix Sum
    if (vec->vector_count > ThreadNr)
    {
        for (int i = 0; i < vec->vector_len; i++)
        {
            thrust::fill(thrust::device, dev_row, dev_row + vec->vector_count, 0u);
            scanLargeDeviceArray(dev_row, dev_vec, vec->vector_count, i);
            thrust::copy(thrust::device, dev_row, dev_row + vec->vector_count, dev_vec + i * vec->vector_count);

            thrust::fill(thrust::device, dev_row, dev_row + vec->vector_count, 0u);
            scanLargeDeviceArray(dev_row, dev_invVec, vec->vector_count, i);
            thrust::copy(thrust::device, dev_row, dev_row + vec->vector_count, dev_invVec + i * vec->vector_count);
        }

    }
    else
    {
        for (int i = 0; i < vec->vector_len; i++)
        {
            scanSmallDeviceArray(dev_row, dev_vec, vec->vector_count, i);
            thrust::copy(thrust::device, dev_row, dev_row + vec->vector_count, dev_vec + i * vec->vector_count);

            scanSmallDeviceArray(dev_row, dev_invVec, vec->vector_count, i);
            thrust::copy(thrust::device, dev_row, dev_row + vec->vector_count, dev_invVec + i * vec->vector_count);
        }

    }


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_temp, start, stop);
    time += time_temp;
    printf("Exclusive SUM time %f [s]\n", time_temp / 1000);


    hipEventRecord(start, 0);

    setRightOrder(dev_vec, dev_perm, dev_row, vec->vector_count, vec->vector_len);
    setRightOrder(dev_invVec, dev_invPerm, dev_row, vec->vector_count, vec->vector_len);

    hipFree(dev_perm);
    hipFree(dev_invPerm);

    //Final table

    create_table(dev_vec, dev_invVec, dev_finalTable, dev_row, vec->vector_count, vec->vector_len);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_temp, start, stop);
    time += time_temp;
    printf("Generating final table %f [s]\n", time_temp / 1000);

    hipFree(dev_vec);
    hipFree(dev_invVec);

    hipFree(dev_row);


    cudaStatus = hipMalloc((void**)&dev_row, vec->vector_len * vec->vector_count * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_perm, vec->vector_len * vec->vector_count * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    blockNr = vec->vector_count * vec->vector_len / ThreadNr;
    if (vec->vector_count * vec->vector_len % ThreadNr != 0)
        blockNr++;


    hipEventRecord(start, 0);
    //Sortowanie finalnej tablicy
    raddixSort(dev_finalTable, dev_perm, dev_row, vec->vector_count * vec->vector_len, 3);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_temp, start, stop);
    time += time_temp;
    printf("Sorting final table %f [s]\n", time_temp / 1000);

    uint32_t* dupa = new uint32_t[vec->vector_count * vec->vector_len *3];

    uint32_t* dupa2 = new uint32_t[vec->vector_count*vec->vector_len];

    cudaStatus = hipMemcpy(dupa,dev_finalTable, vec->vector_count * vec->vector_len*3 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy dupa failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dupa2, dev_perm, vec->vector_count * vec->vector_len * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy dupa failed!");
        goto Error;
    }

    for (int i = 0; i < vec->vector_count * vec->vector_len; i++)
    {
        printf("%d, ", dupa2[i]);
    }
    printf("\n");

    for (int i = 0; i < vec->vector_count * vec->vector_len; i++)
    {
        printf("%d , %d , %d\n", dupa[i], dupa[i + vec->vector_count * vec->vector_len], dupa[i + 2 * vec->vector_count * vec->vector_len]);
    }
    delete[] dupa;
    delete[] dupa2;
    hipEventRecord(start, 0);
    //Zliczanie Par
    
    if (verbose)
    {
        uint32_t* pairsOne;
        uint32_t* pairsTwo;
        cudaStatus = hipMalloc((void**)&pairsOne, 1000*sizeof(uint32_t));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc pairsOne failed!");
            goto Error;
        }

        cudaStatus = hipMemset(pairsOne, 0, 1000*sizeof(uint32_t));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cudaMamset pairOne failed!");
            goto Error;
        }
        cudaStatus = hipMalloc((void**)&pairsTwo, 1000*sizeof(uint32_t));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc pairTwo failed!");
            goto Error;
        }

        cudaStatus = hipMemset(pairsTwo, 0, 1000*sizeof(uint32_t));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cudaMamset pairsTwo failed!");
            goto Error;
        }

        countPairsKernelVerbose << <blockNr, ThreadNr >> > (dev_finalTable, dev_pair_count,dev_perm,pairsOne,pairsTwo, vec->vector_count, vec->vector_len);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_temp, start, stop);
        time += time_temp;
        printf("Finding pairs %f [s]\n", time_temp / 1000);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "findPairsKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching findPairsKernel!\n", cudaStatus);
            goto Error;
        }
        uint32_t* vecOne = new uint32_t[1000];
        uint32_t* vecTwo = new uint32_t[1000];

        cudaStatus = hipMemcpy(vecOne, pairsOne, 1000*sizeof(uint32_t), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy vecOne failed!");
            goto Error;
        }
        cudaStatus = hipMemcpy(vecTwo, pairsTwo, 1000*sizeof(uint32_t), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy vecTwo failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(&pairs, dev_pair_count, sizeof(uint64_t), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }


        printf("Cuda done. Time %f [s]\n", time / 1000);

        cout << "Result. Pairs found: " << pairs << endl;

        int j = 0;
        for (int i = 0; i < 1000; i++)
        {
            if(vecOne[i]!= vecTwo[i])
            {
                printf("%d , %d\n", vecOne[i], vecTwo[i]);
                j++;
            }
            if (j >= 50)
                break;
            
        }
        delete[] vecOne;
        delete[] vecTwo;

        hipFree(pairsOne);
        hipFree(pairsOne);
    }
    else
    {
        countPairsKernel <<<blockNr, ThreadNr >> > (dev_finalTable, dev_pair_count, vec->vector_count, vec->vector_len);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_temp, start, stop);
        time += time_temp;
        printf("Finding pairs %f [s]\n", time_temp / 1000);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "findPairsKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching findPairsKernel!\n", cudaStatus);
            goto Error;
        }


        cudaStatus = hipMemcpy(&pairs, dev_pair_count, sizeof(uint64_t), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }


        printf("Cuda done. Time %f [s]\n", time / 1000);

        cout << "Result. Pairs found: " << pairs / 2 << endl;
    }


Error:
    hipFree(dev_vec);
    hipFree(dev_invVec);
    hipFree(dev_row);
    hipFree(dev_perm);
    hipFree(dev_invPerm);
    hipFree(dev_finalTable);
    hipFree(dev_pair_count);


    return cudaStatus;
}